#include "hip/hip_runtime.h"
#include <CudaThresholding.h>
#include <gkg-processing-container/Volume_i.h>
#include <gkg-communication-cuda/CudaDevices.h>

#include <hip/hip_runtime.h>

#include <iostream>


__global__ void CudaThresholdingKernelFloat( float* c_v,
                                             float* c_o,
                                             float threshold,
                                             int32_t sizeX,
                                             int32_t sizeY,
                                             int32_t sizeZT )
{

  int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  int32_t sliceSize = sizeX * sizeY;
  int32_t z;

  if ( ( ix < sizeX ) && ( iy < sizeY ) )
  {

    int32_t offset = ix + iy * sizeX;

    for ( z = 0; z < sizeZT; z++, offset += sliceSize )
    {

      if ( c_v[ offset ] >= threshold )
      {

        c_o[ offset ] = c_v[ offset ];

      }

    }

  }

}


void gkg::CudaThreshold( gkg::Volume< float >& data, float threshold )
{

  size_t deviceCount = gkg::CudaDevices::getInstance().size();

  if ( deviceCount < 1 )
  {

    std::cerr << "No CUDA compatible device detected" << std::endl;
    return;

  }

  int32_t deviceID = gkg::CudaDevices::getInstance().getBestDeviceId();
  gkg::CudaProperties bestProperties = 
                      gkg::CudaDevices::getInstance().getProperties( deviceID );

  float* v = &data( 0 );
  float* c_v = 0;
  float* c_o = 0;
  int32_t sizeX = data.getSizeX();
  int32_t sizeY = data.getSizeY();
  int32_t sizeZT = data.getSizeZ() * data.getSizeT();
  size_t size = sizeX * sizeY * sizeZT * sizeof( float );
  hipEvent_t start, stop, mem;

  CHECK_CUDA_SUCCESS( hipGetDevice( &deviceID ), "Get device ID" );

  hipDeviceSynchronize();

  CHECK_CUDA_SUCCESS( hipEventCreate( &start ), "Create start event" );
  CHECK_CUDA_SUCCESS( hipEventCreate( &stop ), "Create stop event" );
  CHECK_CUDA_SUCCESS( hipEventCreate( &mem ), "Create mem event" );
  CHECK_CUDA_SUCCESS( hipEventRecord( start, 0 ), "Get start time" );

  CHECK_CUDA_SUCCESS( hipMalloc( &c_v, size ), "Alloc c_v" );
  CHECK_CUDA_SUCCESS( hipMalloc( &c_o, size ), "Alloc c_o" );

  CHECK_CUDA_SUCCESS( hipHostRegister( v, size, hipHostRegisterPortable ),
                      "Host register v" );
  CHECK_CUDA_SUCCESS( hipMemcpy( c_v, v, size, hipMemcpyHostToDevice ),
                      "Memcpy v -> c_v" );
  CHECK_CUDA_SUCCESS( hipMemset( c_o, 0, size ), "Memset c_o to 0" );
  CHECK_CUDA_SUCCESS( hipEventRecord( mem, 0 ), "Get mem time" );

  dim3 Db( 32, 32 );
  dim3 Dg( sizeX / 32 + ( ( sizeX % 32 ) ? 1 : 0 ),
           sizeY / 32 + ( ( sizeY % 32 ) ? 1 : 0 ) );

  CudaThresholdingKernelFloat<<< Dg, Db >>>( c_v,
                                             c_o,
                                             threshold,
                                             sizeX,
                                             sizeY,
                                             sizeZT );

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  CHECK_CUDA_SUCCESS( hipMemcpy( v, c_o, size, hipMemcpyDeviceToHost ),
                      "Memcpy c_o -> v" );
  CHECK_CUDA_SUCCESS( hipHostUnregister( v ), "Host unregister v" );

  float elapsedTime, memTime;

  hipEventElapsedTime( &elapsedTime, start, stop );
  hipEventElapsedTime( &memTime, mem, stop );

  elapsedTime *= 1e-3;
  memTime *= 1e-3;

  hipEventDestroy( mem );
  hipEventDestroy( stop );
  hipEventDestroy( start );
  hipDeviceSynchronize();

  hipFree( c_o );
  hipFree( c_v );

  hipDeviceReset();

  std::cout << "GPU global time = " << elapsedTime << "s" 
            << " ;  computation time = " << memTime << std::endl;

}
